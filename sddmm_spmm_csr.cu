#include "hip/hip_runtime.h"
// System includes
#include <iostream>
// #include <cassert>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Sparse Operations, hipsparseSpMM and hipsparseSDDMM
#include <hipsparse.h>

using namespace std;

// Class for Matrix Multiplication Components
class MatMul {
public:
    int A_num_rows, A_num_cols, B_num_rows, B_num_cols, C_nnz, lda, ldb, A_size, B_size, *hC_offsets, *hC_columns;
    float *matA, *matB, *hC_values, *hC_result, alpha, beta;

    MatMul(float *matA_in, int A_num_rows_in, int A_num_cols_in, int A_size_in, int lda_in, float *matB_in,
           int B_num_rows_in, int B_num_cols_in, int B_size_in, int ldb_in, int C_nnz_in, int *hC_offsets_in,
           int *hC_columns_in, float *hC_values_in, float *hC_result_in, float alpha_in, float beta_in) {

        if (A_num_cols_in != B_num_rows_in) {
            throw std::invalid_argument("Number of columns in matrix A does not equal number of rows in matrix B.");
        }
        matA = matA_in;
        A_num_rows = A_num_rows_in;
        A_num_cols = A_num_cols_in;
        A_size = A_size_in;
        lda = lda_in;

        matB = matB_in;
        B_num_rows = B_num_rows_in;
        B_num_cols = B_num_cols_in;
        B_size = B_size_in;
        ldb = ldb_in;

        C_nnz = C_nnz_in;
        hC_offsets = hC_offsets_in;
        hC_columns = hC_columns_in;
        hC_values = hC_values_in;
        hC_result = hC_result_in;

        alpha = alpha_in;
        beta = beta_in;
    }
};

void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

void calcPerf(float msecTotal, int nIter, int A_rows, int A_cols, int B_rows) {
    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(A_rows) *
                               static_cast<double>(A_cols) *
                               static_cast<double>(B_rows);
    double gigaFlops =
            (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
           gigaFlops, msecPerMatrixMul, flopsPerMatrixMul);
}

bool validate(int resultRows, int resultCols, float *h_C, int A_rows) {
    float valB = 0.01f;
    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6; // machine zero

    for (int i = 0; i < static_cast<int>(resultRows * resultCols); i++) {
        double abs_err = fabs(h_C[i] - (A_rows * valB));
        double dot_length = A_rows;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i,
                   h_C[i], A_rows * valB, eps);
            correct = false;
        }
    }
    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");
    return correct;
}

// Kernel
bool _sddmm(MatMul input, hipStream_t stream) {
    //--------------------------------------------------------------------------
    // Device memory management
    int *dC_offsets, *dC_columns;
    float *dC_values, *dB, *dA;
    checkCudaErrors(hipMalloc((void **) &dA, input.A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &dB, input.B_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &dC_offsets,
                               (input.A_num_rows + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &dC_columns, input.C_nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &dC_values, input.C_nnz * sizeof(float)));

    checkCudaErrors(hipMemcpy(dA, input.matA, input.A_size * sizeof(float),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dB, input.matB, input.B_size * sizeof(float),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dC_offsets, input.hC_offsets,
                               (input.A_num_rows + 1) * sizeof(int),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dC_columns, input.hC_columns, input.C_nnz * sizeof(int),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dC_values, input.hC_values, input.C_nnz * sizeof(float),
                               hipMemcpyHostToDevice));
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    checkCudaErrors(hipsparseCreate(&handle));
    // Create dense matrix A
    checkCudaErrors(hipsparseCreateDnMat(&matA, input.A_num_rows, input.A_num_cols, input.lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // Create dense matrix B
    checkCudaErrors(hipsparseCreateDnMat(&matB, input.A_num_cols, input.B_num_cols, input.ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // Create sparse matrix C in CSR format
    checkCudaErrors(hipsparseCreateCsr(&matC, input.A_num_rows, input.B_num_cols, input.C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    // allocate an external buffer if needed
    checkCudaErrors(hipsparseSDDMM_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
            HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize));
    checkCudaErrors(hipMalloc(&dBuffer, bufferSize));

    // execute preprocess (optional)
    checkCudaErrors(hipsparseSDDMM_preprocess(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
        HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer));
    // execute SpMM
    checkCudaErrors(hipsparseSDDMM(
          handle,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
          HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer));
    // destroy matrix/vector descriptors
    checkCudaErrors(hipsparseDestroyDnMat(matA));
    checkCudaErrors(hipsparseDestroyDnMat(matB));
    checkCudaErrors(hipsparseDestroySpMat(matC));
    checkCudaErrors(hipsparseDestroy(handle));
    //--------------------------------------------------------------------------
    // device result check
    // Copy result from device to host
    checkCudaErrors(
            hipMemcpyAsync(input.hC_values, dC_values, input.C_nnz * sizeof(float),
                            hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    // B rows = C rows && A || B cols = C cols
    bool correct = validate(input.B_num_rows, input.B_num_cols, input.hC_values, input.A_num_rows);
    //--------------------------------------------------------------------------
    // device memory deallocation
    checkCudaErrors(hipFree(dBuffer));
    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dB));
    checkCudaErrors(hipFree(dC_offsets));
    checkCudaErrors(hipFree(dC_columns));
    checkCudaErrors(hipFree(dC_values));
    return correct;
}

void _spmm(MatMul input, hipStream_t stream) {

}

bool _sddmmSpmm(MatMul input, hipStream_t stream) {
    bool correct = _sddmm(input, stream);
    if(!correct) {
        printf("sddmm failed");
    }
    return correct;
}

// Main
int main() {
    // Testing Variables -> from sddmm_csr_example.c
    int A_num_rows = 4;
    int A_num_cols = 4;
    int B_num_rows = A_num_cols;
    int B_num_cols = 3;
    int C_nnz = 9;
    int lda = A_num_cols;
    int ldb = B_num_cols;
    int A_size = lda * A_num_rows;
    int B_size = ldb * B_num_rows;
    float hA[] = {1.0f, 2.0f, 3.0f, 4.0f,
                  5.0f, 6.0f, 7.0f, 8.0f,
                  9.0f, 10.0f, 11.0f, 12.0f,
                  13.0f, 14.0f, 15.0f, 16.0f};
    float hB[] = {1.0f, 2.0f, 3.0f,
                  4.0f, 5.0f, 6.0f,
                  7.0f, 8.0f, 9.0f,
                  10.0f, 11.0f, 12.0f};
    int hC_offsets[] = {0, 3, 4, 7, 9};
    int hC_columns[] = {0, 1, 2, 1, 0, 1, 2, 0, 2};
    float hC_values[] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                         0.0f, 0.0f, 0.0f, 0.0f};
    float hC_result[] = {70.0f, 80.0f, 90.0f,184.0f,246.0f,
                         288.0f, 330.0f,334.0f, 450.0f};
    float alpha = 1.0f;
    float beta = 0.0f;

    MatMul test = MatMul(
        hA,
        A_num_rows,
        A_num_cols,
        A_size,
        lda,
        hB,
        B_num_rows,
        B_num_cols,
        B_size,
        ldb,
        C_nnz,
        hC_offsets,
        hC_columns,
        hC_values,
        hC_result,
        alpha,
        beta
    );

    printf("Computing result using CUDA Kernel...\n");

    // Initialize timing variables
    hipStream_t stream;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // Performs warmup operation using _sddmmSpmm CUDA kernel
    bool correct = _sddmmSpmm(test, stream);

    printf("done\n");
    checkCudaErrors(hipStreamSynchronize(stream));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, stream));

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++) {
        _sddmmSpmm(test, stream);
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    calcPerf(msecTotal, nIter, test.A_num_rows, test.A_num_cols, test.B_num_rows);

    // old correct code

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}