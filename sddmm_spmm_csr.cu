// System includes
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Sparse Operations, cusparseSpMM and cusparseSDDMM
#include <hipsparse.h>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}


using namespace std;

// Class for Matrix Multiplication Components
class MatMul {
public:
    int A_num_rows, A_num_cols, B_num_rows, B_num_cols, C_nnz, lda, ldb, A_size, B_size, *hC_offsets, *hC_columns,
    D_num_rows, D_num_cols, D_size, ldd, lde, E_size;
    float *matA, *matB, *hC_values, *hC_result, alpha, beta, *matD, *matE;

    MatMul(float *matA_in, int A_num_rows_in, int A_num_cols_in, int A_size_in, int lda_in, float *matB_in,
           int B_num_rows_in, int B_num_cols_in, int B_size_in, int ldb_in, int C_nnz_in, int *hC_offsets_in,
           int *hC_columns_in, float *hC_values_in, float *hC_result_in, float alpha_in, float beta_in, float *D_mat_in,
           int D_num_rows_in, int D_num_cols_in, int D_size_in, int ldd_in, float *matE_in, int lde_in, int E_size_in) {

        if (A_num_cols_in != B_num_rows_in) {
            throw std::invalid_argument("Number of columns in matrix A does not equal number of rows in matrix B.");
        }
        matA = matA_in;
        A_num_rows = A_num_rows_in;
        A_num_cols = A_num_cols_in;
        A_size = A_size_in;
        lda = lda_in;

        matB = matB_in;
        B_num_rows = B_num_rows_in;
        B_num_cols = B_num_cols_in;
        B_size = B_size_in;
        ldb = ldb_in;

        C_nnz = C_nnz_in;
        hC_offsets = hC_offsets_in;
        hC_columns = hC_columns_in;
        hC_values = hC_values_in;
        hC_result = hC_result_in;

        matD = D_mat_in;
        D_num_rows = D_num_rows_in;
        D_num_cols = D_num_cols_in;
        D_size = D_size_in;
        ldd = ldd_in;

        matE = matE_in;
        lde = lde_in;
        E_size = E_size_in;

        alpha = alpha_in;
        beta = beta_in;
    }
};

void calcPerf(float msecTotal, int nIter, int A_rows, int A_cols, int B_rows) {
    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(A_rows) *
                               static_cast<double>(A_cols) *
                               static_cast<double>(B_rows);
//    double gigaFlops =
//            (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    double flops =
            (flopsPerMatrixMul) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f Flop/s, Time= %.3f msec, Size= %.0f Ops\n",
           flops, msecPerMatrixMul, flopsPerMatrixMul);
}

bool validate_sddmm(int C_nnz, float *hC_values, float *hC_result) {
    int correct = 1;
    for (int i = 0; i < C_nnz; i++) {
        if (hC_values[i] != hC_result[i]) {
            correct = 0; // direct floating point comparison is not reliable
            break;
        }
    }
    if (correct)
        printf("sddmm_csr_example test PASSED\n");
    else
        printf("sddmm_csr_example test FAILED: wrong result\n");
    return correct;
}

bool validate_spmm(int A_num_rows, int B_num_cols, float *hC_values, float *hC_result) {
    int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        for (int j = 0; j < B_num_cols; j++) {
            if (hC_values[i + j * A_num_rows] != hC_result[i + j * A_num_rows]) {
                correct = 0; // direct floating point comparison is not reliable
                break;
            }
        }
    }
    if (correct)
        printf("spmm_csr_example test PASSED\n");
    else
        printf("spmm_csr_example test FAILED: wrong result\n");
    return correct;
}

// Kernel
int _sddmm(MatMul input, hipStream_t stream) {
    //--------------------------------------------------------------------------
    // Device memory management
    int *dC_offsets, *dC_columns;
    float *dC_values, *dB, *dA;
    CHECK_CUDA(hipMalloc((void **) &dA, input.A_size * sizeof(float)))
    CHECK_CUDA(hipMalloc((void **) &dB, input.B_size * sizeof(float)))
    CHECK_CUDA(hipMalloc((void **) &dC_offsets,
                               (input.A_num_rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **) &dC_columns, input.C_nnz * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **) &dC_values, input.C_nnz * sizeof(float)))

    CHECK_CUDA(hipMemcpy(dA, input.matA, input.A_size * sizeof(float),
                               hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dB, input.matB, input.B_size * sizeof(float),
                               hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dC_offsets, input.hC_offsets,
                               (input.A_num_rows + 1) * sizeof(int),
                               hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dC_columns, input.hC_columns, input.C_nnz * sizeof(int),
                               hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dC_values, input.hC_values, input.C_nnz * sizeof(float),
                               hipMemcpyHostToDevice))
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t handle = nullptr;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void *dBuffer = nullptr;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    // Create dense matrix A
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matA, input.A_num_rows, input.A_num_cols, input.lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW))
    // Create dense matrix B
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, input.A_num_cols, input.B_num_cols, input.ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW))
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(&matC, input.A_num_rows, input.B_num_cols, input.C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSDDMM_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
            HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize))
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

    // execute preprocess (optional)
    CHECK_CUSPARSE(hipsparseSDDMM_preprocess(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
        HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer))
    // execute SpMM
    CHECK_CUSPARSE(hipsparseSDDMM(
          handle,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
          HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer))
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
    CHECK_CUSPARSE(hipsparseDestroy(handle))
    //--------------------------------------------------------------------------
    // device result check
    // Copy result from device to host
    CHECK_CUDA(
            hipMemcpyAsync(input.hC_values, dC_values, input.C_nnz * sizeof(float),
                            hipMemcpyDeviceToHost, stream))
    CHECK_CUDA(hipStreamSynchronize(stream))

    // B rows = C rows && A || B cols = C cols
    bool correct = validate_sddmm(input.C_nnz, input.hC_values, input.hC_result);
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA(hipFree(dBuffer))
    CHECK_CUDA(hipFree(dA))
    CHECK_CUDA(hipFree(dB))
    CHECK_CUDA(hipFree(dC_offsets))
    CHECK_CUDA(hipFree(dC_columns))
    CHECK_CUDA(hipFree(dC_values))
    return EXIT_SUCCESS;
}

int _spmm(MatMul input, hipStream_t stream) {
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dB, *dC;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (input.A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, input.C_nnz * sizeof(int))    )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  input.C_nnz * sizeof(float))  )
    CHECK_CUDA( hipMalloc((void**) &dB, input.D_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC, input.E_size * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, input.hC_offsets,
                           (input.A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, input.hC_columns, input.C_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, input.hC_values, input.C_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, input.matD, input.D_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, input.matE, input.E_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, input.A_num_rows, input.B_num_cols, input.C_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, input.B_num_cols, input.D_num_cols, input.ldd, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, input.A_num_rows, input.D_num_cols, input.lde, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(
            handle,
         HIPSPARSE_OPERATION_NON_TRANSPOSE,
         HIPSPARSE_OPERATION_NON_TRANSPOSE,
         &input.alpha, matA, matB, &input.beta, matC, HIP_R_32F,
         HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(input.matE, dC, input.E_size * sizeof(float),
                           hipMemcpyDeviceToHost) )
    int correct = 1;
    for (int i = 0; i < input.A_num_rows; i++) {
        for (int j = 0; j < input.D_num_cols; j++) {
//            if (input.matE[i + j * input.lde] != hC_result[i + j * ldc]) {
//                correct = 0; // direct floating point comparison is not reliable
//                break;
//            }
            printf("%f ",input.matE[i+j * input.lde]);
        }
        printf("\n");
    }
    if (correct)
        printf("spmm_csr_example test PASSED\n");
    else
        printf("spmm_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
    return EXIT_SUCCESS;
}

int _sddmmSpmm(MatMul input, hipStream_t stream) {
    _sddmm(input, stream);
    _spmm(input, stream);
    return EXIT_SUCCESS;
}

// Main
int main() {

    // Testing Variables -> from sddmm_csr_example.c
    int A_num_rows = 4;
    int A_num_cols = 4;
    int B_num_rows = A_num_cols;
    int B_num_cols = 3;
    int C_nnz = 9;
    int lda = A_num_cols;
    int ldb = B_num_cols;
    int A_size = lda * A_num_rows;
    int B_size = ldb * B_num_rows;

    int D_num_rows = B_num_cols;
    int D_num_cols = 4;
    int ldd = D_num_rows;
    int D_size = ldd * D_num_cols;

    float hA[] = {1.0f, 2.0f, 3.0f, 4.0f,
                  5.0f, 6.0f, 7.0f, 8.0f,
                  9.0f, 10.0f, 11.0f, 12.0f,
                  13.0f, 14.0f, 15.0f, 16.0f};
    float hB[] = {1.0f, 2.0f, 3.0f,
                  4.0f, 5.0f, 6.0f,
                  7.0f, 8.0f, 9.0f,
                  10.0f, 11.0f, 12.0f};
    float hD[] = { 1.0f,  2.0f,  3.0f,  4.0f,
                   5.0f,  6.0f,  7.0f,  8.0f,
                   9.0f, 10.0f, 11.0f, 12.0f};

    int hC_offsets[] = {0, 3, 4, 7, 9};
    int hC_columns[] = {0, 1, 2, 1, 0, 1, 2, 0, 2};
    float hC_values[] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                         0.0f, 0.0f, 0.0f, 0.0f};
    float hC_result[] = {70.0f, 80.0f, 90.0f,184.0f,246.0f,
                         288.0f, 330.0f,334.0f, 450.0f};

    int lde = A_num_rows;
    int E_size = lde * D_num_cols;
    float hE[] = { 0.0f, 0.0f, 0.0f, 0.0f,
                   0.0f, 0.0f, 0.0f, 0.0f,
                   0.0f, 0.0f, 0.0f, 0.0f,
                   0.0f, 0.0f, 0.0f, 0.0f };
    float alpha = 1.0f;
    float beta = 0.0f;

    MatMul test = MatMul(
        hA,
        A_num_rows,
        A_num_cols,
        A_size,
        lda,
        hB,
        B_num_rows,
        B_num_cols,
        B_size,
        ldb,
        C_nnz,
        hC_offsets,
        hC_columns,
        hC_values,
        hC_result,
        alpha,
        beta,
        hD,
        D_num_rows,
        D_num_cols,
        D_size,
        ldd,
        hE,
        lde,
        E_size
    );

    printf("Computing result using CUDA Kernel...\n");

    // Initialize timing variables
    hipStream_t stream;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start))
    CHECK_CUDA(hipEventCreate(&stop))
    CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking))

    printf("done\n");
    CHECK_CUDA(hipStreamSynchronize(stream))

    // Execute the kernel
    int nIter = 300;

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, stream))

    for (int j = 0; j < nIter; j++) {
        _sddmmSpmm(test, stream);
    }

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, stream))

    // Wait for the stop event to complete
    CHECK_CUDA(hipEventSynchronize(stop))

    float msecTotal = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&msecTotal, start, stop))

    calcPerf(msecTotal, nIter, test.A_num_rows, test.A_num_cols, test.B_num_rows);

    return EXIT_SUCCESS;
}