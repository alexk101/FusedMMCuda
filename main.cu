#include <iostream>

int main() {
    hipError_t err = hipMalloc((void**) &dA, A_size * sizeof(float));
    CHECK_CUDA(err, __LINE__-1);
    err = hipMalloc((void**) &dB, B_size * sizeof(float));
    std::cout << "Hello, World!" << std::endl;
    return 0;
}
